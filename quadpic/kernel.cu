#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define rows 8//image rows
#define columns 8//image columns
#define noOfThreadsInBlock 4
#define noOfBlocks 4
#define noOfThreadInBlockBy2 noOfThreadsInBlock/2
#define gridDim3 dim3(noOfBlocks/2, noOfBlocks/2)
#define blockDim3 dim3(noOfThreadInBlockBy2, noOfThreadInBlockBy2)

hipError_t generateOutputWithCuda(int *c, const int *a, unsigned int size);

__device__ int getAbsoluteIndex(int threadIndexX, int threadIndexY) {
	int xIndexInGrid = blockIdx.x*blockDim.x + threadIndexX;//global x in the grid
	int yIndexInGrid = blockIdx.y*blockDim.y + threadIndexY;//global y in the grid

	//TODO add to global x and global y in the whole image
	return xIndexInGrid + yIndexInGrid*rows;
}

__global__ void getAverageKernel(const int *a, int *mutex, float* average, int quadNo)
{
	__shared__ int sum[noOfThreadInBlockBy2];
	int index = getAbsoluteIndex(threadIdx.x, threadIdx.y);
	int threadIndex = threadIdx.x + threadIdx.y*blockDim.x;//local 1d index into the block
	int localKernelNoOfThreadsHalf = (blockDim.x * blockDim.y) / 2;
	unsigned int i = localKernelNoOfThreadsHalf;
	while (i != 0) {
		if (threadIndex < i) {
			if (i == localKernelNoOfThreadsHalf) {//first iteration
				int x2 = (threadIdx.x + i) % blockDim.x;
				int y2 = (threadIdx.y + i) / blockDim.x;
				int index2 = getAbsoluteIndex(x2, y2);

				sum[threadIndex] = a[index] + a[index2];
			}
			else {
				sum[threadIndex] += sum[threadIndex + i];
			}
			
		}

		__syncthreads();
		i /= 2;
	}

	if (threadIndex == 0) {
		//TODO atomic update in global
		//int blockIndex = blockIdx.x + blockIdx.y*blockDim.x;
		//c[blockIndex] = sum[0];
		while(atomicCAS(mutex+quadNo,0,1) != 0);  //lock
		average[quadNo] += (float) sum[0]/(gridDim.x*gridDim.y*blockDim.x*blockDim.y);//TODO will have to account for total pixels in this quad (striding and all)
		atomicExch(mutex+quadNo, 0);  //unlock
	}
}

__global__ void kernelToRuleThemAll(int *c, const int *a, int *mutex, float* average) {
	getAverageKernel <<<gridDim3, blockDim3>>>(a, mutex, average, 0);

	//have to parallelize in streams the different kernel launches
	//one stream will calc average then calc deviation
	//have to use hipDeviceSynchronize() after deviation calculations to get minimum from all
	//paint after all iterations are done
}

int main()
{
    const int arraySize = rows*columns;
    /*const int imageData[arraySize] = { 0, 1,	4, 5,
									   2, 3,	6, 7,	
									   
									   8, 9,	12, 13,
									   10, 11,	14, 15,
									};*/

	int imageData[64];
	int id;
	for (id = 0; id < 64; id++) {
		imageData[id] = id;
	}
    int c[noOfBlocks] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = generateOutputWithCuda(c, imageData, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t generateOutputWithCuda(int *c, const int *a, unsigned int size)
{
    int *dev_a = 0;
    int *dev_c = 0;
	int *dev_mutex = 0;
	float *dev_average = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_mutex, 4 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_average, 4 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU.
	kernelToRuleThemAll << <1, 1 >> >(dev_c, dev_a, dev_mutex, dev_average);
    //getAverageKernel<<<gridDim3, blockDim3>>>(dev_c, dev_a, dev_mutex, dev_average, 0);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, noOfBlocks * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	float avg[4] = { 0 };
	float *average = avg;
	cudaStatus = hipMemcpy(average, dev_average, 4 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	printf("Aver %f", avg[0]);

	/*int i = 0;
	for (i = 0; i < noOfBlocks; i++) {
		printf("\nOutput %d", c[i]);
	}*/

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    
    return cudaStatus;
}

/*void checkError(hipError_t err, char * errorString) {
	if (err != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(dev_c);
		hipFree(dev_a);

		exit(1);
	}
}*/

//one kernel to rule them all, app store near you
