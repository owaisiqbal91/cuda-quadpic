#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define noOfIterations 2500
#define noOfQuads noOfIterations*3

#define rows 8//image rows
#define columns 8//image columns
#define noOfThreadsInBlock 16
//#define noOfBlocks 4
#define noOfThreadInBlockBy2 noOfThreadsInBlock/2
//#define gridDim3 dim3(noOfBlocks/2, noOfBlocks/2)
//#define blockDim3 dim3(noOfThreadInBlockBy2, noOfThreadInBlockBy2)

hipError_t generateOutputWithCuda(int *c, const int *a, unsigned int size);

__device__ int getAbsoluteIndex(int threadIndexX, int threadIndexY, int quadStartX, int quadStartY) {
	int xIndexInGrid = blockIdx.x*blockDim.x + threadIndexX;//global x in the grid
	int yIndexInGrid = blockIdx.y*blockDim.y + threadIndexY;//global y in the grid

	xIndexInGrid += quadStartX;
	yIndexInGrid += quadStartY;

	return xIndexInGrid + yIndexInGrid*rows;
}

__global__ void getAverageKernel(const int *a, int *mutex, float* average, int quadNo, int quadStartX, int quadStartY, int quadEndX, int quadEndY)
{
	int threadAbsX = (threadIdx.x + blockDim.x*blockIdx.x) + quadStartX;
	int threadAbsY = (threadIdx.y + blockDim.y*blockIdx.y) + quadStartY;

	if (threadAbsX <= quadEndX && threadAbsY <= quadEndY /*&& blockIdx.x == 0 && blockIdx.y == 0*/) {
		__shared__ int sum[noOfThreadInBlockBy2];
		int index = getAbsoluteIndex(threadIdx.x, threadIdx.y, quadStartX, quadStartY);
		int threadIndex = threadIdx.x + threadIdx.y*blockDim.x;//local 1d index into the block
		int localKernelNoOfThreadsHalf = (blockDim.x * blockDim.y) / 2;
		unsigned int i = localKernelNoOfThreadsHalf;
		while (i != 0) {
			if (threadIndex < i) {
				if (i == localKernelNoOfThreadsHalf) {//first iteration

					sum[threadIndex] = a[index];
					//printf("\nSum[%d] = %d", threadIndex, index);
					int x2 = threadIdx.x;// + i) % blockDim.x;
					int y2 = threadIdx.y + (i / blockDim.x);
					int threadAbsX2 = (x2 + blockDim.x*blockIdx.x) + quadStartX;
					int threadAbsY2 = (y2 + blockDim.y*blockIdx.y) + quadStartY;
					if (threadAbsX2 <= quadEndX && threadAbsY2 <= quadEndY) {
						int index2 = getAbsoluteIndex(x2, y2, quadStartX, quadStartY);

						sum[threadIndex] += a[index2];
						//printf("\nSum[%d] 2nd = %d, threadIdy: %d, i: %d, blockDimy: %d, calc: %d, y2: %d", threadIndex, index2, threadIdx.y, i, blockDim.x, (i+threadIdx.y)/blockDim.x, y2);
					}
				}
				else {
					sum[threadIndex] += sum[threadIndex + i];
				}

			}

			__syncthreads();
			i /= 2;
		}

		if (threadIndex == 0) {
			//int blockIndex = blockIdx.x + blockIdx.y*blockDim.x;
			while (atomicCAS(mutex + quadNo, 0, 1) != 0);  //lock
			printf("\nBlock sum is %d", sum[0]);
			average[quadNo] += (float)sum[0] / ((quadEndX - quadStartX + 1)*(quadEndY - quadStartY + 1));
			atomicExch(mutex + quadNo, 0);  //unlock
		}
	}
}

__global__ void getScoreAndPaintKernel(const int *a, int *mutex, float* average, float* score, int quadNo, int quadStartX, int quadStartY, int quadEndX, int quadEndY) {

	int threadAbsX = (threadIdx.x + blockDim.x*blockIdx.x) + quadStartX;
	int threadAbsY = (threadIdx.y + blockDim.y*blockIdx.y) + quadStartY;

	if (threadAbsX <= quadEndX && threadAbsY <= quadEndY) {
		__shared__ float error[noOfThreadInBlockBy2];
		int index = getAbsoluteIndex(threadIdx.x, threadIdx.y, quadStartX, quadStartY);
		int threadIndex = threadIdx.x + threadIdx.y*blockDim.x;//local 1d index into the block
		int localKernelNoOfThreadsHalf = (blockDim.x * blockDim.y) / 2;
		unsigned int i = localKernelNoOfThreadsHalf;
		float avg = average[quadNo];
		while (i != 0) {
			if (threadIndex < i) {
				if (i == localKernelNoOfThreadsHalf) {//first iteration

					error[threadIndex] = pow(a[index] - avg, 2);

					int x2 = threadIdx.x;// + i) % blockDim.x;
					int y2 = threadIdx.y + (i / blockDim.x);
					int threadAbsX2 = (x2 + blockDim.x*blockIdx.x) + quadStartX;
					int threadAbsY2 = (y2 + blockDim.y*blockIdx.y) + quadStartY;
					if (threadAbsX2 <= quadEndX && threadAbsY2 <= quadEndY) {
						int index2 = getAbsoluteIndex(x2, y2, quadStartX, quadStartY);
						error[threadIndex] += pow(a[index2] - avg, 2);
					}
				}
				else {
					error[threadIndex] += error[threadIndex + i];
				}

			}

			__syncthreads();
			i /= 2;
		}

		if (threadIndex == 0) {
			while (atomicCAS(mutex + quadNo, 0, 1) != 0);  //lock
			score[quadNo] += error[0] / ((quadEndX - quadStartX + 1)*(quadEndY - quadStartY + 1));
			atomicExch(mutex + quadNo, 0);  //unlock
		}
	}
}

__global__ void getMaxScoreKernel(float* globalScores, int* maxScoreIndex, int currentTotalQuads) {
	__shared__ float maxValues[512];
	__shared__ int maxIndex[512];

	int localKernelNoOfThreadsHalf = blockDim.x / 2;
	unsigned int i = localKernelNoOfThreadsHalf;

	int threadGlobalIndex = blockIdx.x*blockDim.x + threadIdx.x;

	if (threadGlobalIndex < currentTotalQuads) {
		while (i != 0) {
			if (threadIdx.x < i) {
				if (i == localKernelNoOfThreadsHalf) {//first iteration

					maxValues[threadIdx.x] = globalScores[threadGlobalIndex];
					maxIndex[threadIdx.x] = threadGlobalIndex;

					int threadGlobalIndex2 = threadGlobalIndex + i;
					
					if (threadGlobalIndex2 < currentTotalQuads) {
						if (globalScores[threadGlobalIndex2] > maxValues[threadIdx.x]) {
							maxValues[threadIdx.x] = globalScores[threadGlobalIndex2];
							maxIndex[threadIdx.x] = threadGlobalIndex2;
						}
					}
				}
				else {
					if (maxValues[threadIdx.x + i] > maxValues[threadIdx.x]) {
						maxValues[threadIdx.x] = maxValues[threadIdx.x + i];
						maxIndex[threadIdx.x] = maxIndex[threadIdx.x + i];
					}
				}
			}

			__syncthreads();
			i /= 2;
		}

		if (threadIdx.x == 0) {
			printf("\nMax value is %f, %d", maxValues[0], maxIndex[0]);
		}
	}
}

__device__ struct quad {
	short int startX;
	short int startY;
	short int endX;
	short int endY;
};


__global__ void kernelToRuleThemAll(int *c, const int *a, int *mutex, float* average, float *score, float* globalScores, int* maxScoreIndex) {

	quad quads[noOfQuads];

	hipStream_t s1;
	hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);

	//TODO shared memory size
	//getAverageKernel <<<dim3(4,4), dim3(4,4), 0, s1>>>(a, mutex, average, 1, 0, 0, 7, 7);
	//getScoreAndPaintKernel << <dim3(4, 4), dim3(4,4), 0, s1>> >(a, mutex, average, score, 1, 0, 0, 7, 7);
	//TODO remember to square root error

	//find max
	globalScores[0] = 1;
	globalScores[1] = 2;
	globalScores[2] = 3;
	globalScores[3] = 4;
	globalScores[4] = 5;
	globalScores[5] = 6;
	getMaxScoreKernel<<<1, 8>>>(globalScores, maxScoreIndex, 6);

	/*hipDeviceSynchronize();
	printf("\nAverage is %f", average[1]);
	printf("\nScore is %f", score[1]);*/
}

int main()
{
	/*size_t free, total;
	hipMemGetInfo(&free,
		&total
	);
	printf("\nCUDA FREE: %zu, TOTAL %zu", free, total);*/

    const int arraySize = rows*columns;
    /*const int imageData[arraySize] = { 0, 1,	4, 5,
									   2, 3,	6, 7,	
									   
									   8, 9,	12, 13,
									   10, 11,	14, 15,
									};*/

	int imageData[64];
	int id;
	for (id = 0; id < 64; id++) {
		imageData[id] = id;
	}
    int c[rows*columns] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = generateOutputWithCuda(c, imageData, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "generateOutputWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t generateOutputWithCuda(int *c, const int *a, unsigned int size)
{
    int *dev_a = 0;
    int *dev_c = 0;
	int *dev_mutex = 0;
	float *dev_average = 0;
	float *dev_score = 0;
	float *dev_globalScores = 0;
	int *dev_maxScoreIndex = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_mutex, 4 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_average, 4 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_score, 4 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_globalScores, noOfQuads * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_maxScoreIndex, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU.
	kernelToRuleThemAll << <1, 1 >> >(dev_c, dev_a, dev_mutex, dev_average, dev_score, dev_globalScores, dev_maxScoreIndex);
    //getAverageKernel<<<gridDim3, blockDim3>>>(dev_c, dev_a, dev_mutex, dev_average, 0);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Main kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernelToRuleThemAll!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, rows*columns * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	/*float avg[4] = { 0 };
	float *average = avg;
	cudaStatus = hipMemcpy(average, dev_average, 4 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}*/

	/*int i = 0;
	for (i = 0; i < noOfBlocks; i++) {
		printf("\nOutput %d", c[i]);
	}*/

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    
    return cudaStatus;
}

//one kernel to rule them all, app store near you
// reduction, dynamic parallelism, streams
